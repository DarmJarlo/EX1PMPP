#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>

// NOTE: if you include stdio.h, you can use printf inside your kernel

#include "common.h"
#include "matrix.h"
#include "mul_gpu.h"

// TODO (Task 4): Implement matrix multiplication CUDA kernel
__global__ void kernel(float* pfMatrixA, float* pfMatrixB, float* pfMatrixC, int m, int n, int k)
{
    
void matrix_mul_gpu(const GPUMatrix &m, const GPUMatrix &n, GPUMatrix &p)
{
t:
	BLOCK_SIZE = m.height * n.width
	dim3 threads(BLOCK_SIZE)
	dim3 grid(GW /BLOCK_SIZE)
	kernel<<<1,threads>>>(m,n)	// TODO (Task 4): Determine execution configuration and call CUDA kernel
}
